#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>

#include "leergml.cuh"


//RED red;
int twom;                // Va a ser el doble del número de aristas
                         // lo queremos para muchas cosas.
int **m;                 // El conteo de las aristas.

int arista_1, arista_2;

__host__ __device__ int kronecker(int x, int y){
    if(x==y) return 1;
    else return 0;
}

__device__ RED dev_red;

void FillStructs(){
    red h_red;

    red *d_red;
    hipGetSymbolAddress( (void**)d_red, dev_red);

    hipMemcpy(d_red, &h_red, sizeof(red), hipMemcpyHostToDevice);
}

__global__ void generar(int i, int j, int k, int l, int * matriz){
    int d_arista_1 = threadIdx.x;
    int d_arista_2 = threadIdx.y;
    d_arista_1 = 0;
    for(i=0;i<red.nnodos;i++){
        for(j=0;j<red.nodo[i].grado;j++){
            d_arista_2=0;
            for(k=0;k<red.nnodos;k++){
                for(l=0;l<red.nodo[k].grado;l++){
                        matriz[d_arista_1+d_arista_2] = \
                            kronecker(red.nodo[k].id,red.nodo[i].arista[j].entrada)* \
                            (1-kronecker(red.nodo[i].id,red.nodo[k].arista[l].entrada));
                    d_arista_2++;
                }
            }
            d_arista_1++;
        }
    }
}
    

int main(int argc, char *argv[])
{
    int u,i,j,k,l;

    fprintf(stderr,"Leyendo la red...\n");

    leer_red(&red,stdin);
    for (u=twom=0; u<red.nnodos; u++) twom += red.nodo[u].grado;
    fprintf(stderr,"Red con %i nodos y %i aristas\n",
        red.nnodos,twom/2);
    fprintf(stderr,"\n");

    int ** nbm;
    int renglon, columna;

    nbm = (int **)calloc(2*twom,sizeof(int));
    for (renglon=0; renglon<2*twom;renglon++){
        nbm[renglon] = (int *)calloc(2*twom,sizeof(int));
    }

    int (*d_nbm)[2*twom];

    hipMalloc((void**)&d_nbm, (4*twom*twom)*sizeof(int));

    hipMemcpy(d_nbm, nbm, (4*twom*twom)*sizeof(int), hipMemcpyHostToDevice);

    int *d_i,*d_j,*d_k,*d_l;

    hipMalloc((void**)&d_i,sizeof(int));
    hipMalloc((void**)&d_j,sizeof(int));
    hipMalloc((void**)&d_k,sizeof(int));
    hipMalloc((void**)&d_l,sizeof(int));

    hipMemcpy(d_i,&i,sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(d_j,&j,sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(d_k,&k,sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(d_l,&l,sizeof(int),hipMemcpyHostToDevice);

    generar<<<2,4>>>(*d_i,*d_j,*d_k,*d_l,*d_nbm);

    for(renglon=0;renglon<twom;renglon++){
        for(columna=0;columna<twom;columna++){
            printf("%i    ", nbm[renglon][columna]);
        }
    printf("\n");
    }
 
    hipFree(d_nbm);
    hipFree(d_i);
    hipFree(d_j);
    hipFree(d_k);
    hipFree(d_l);

    return 0;

}    
