#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <hip/hip_runtime.h>

//#include "leergml.h"
extern "C"{
#include "leergml.h"
}


extern "C"{
int leer_red(RED *red, FILE *stream);
}

//RED red;
int twom;                // Va a ser el doble del número de aristas
                         // lo queremos para muchas cosas.
int **m;                 // El conteo de las aristas.

int arista_1, arista_2;

__host__ __device__ int kronecker(int x, int y){
    if(x==y) return 1;
    else return 0;
}

__device__ RED red;
RED h_red;

void FillStructs(){
    RED h_red;

    RED *d_red;

    hipGetSymbolAddress( (void**)&d_red, red);

    hipMemcpy(d_red, &h_red, sizeof(RED), hipMemcpyHostToDevice);
}

__global__ void generar(int i, int j, int k, int l, int * matriz){
    int d_arista_1 = threadIdx.x;
    int d_arista_2 = threadIdx.y;
    d_arista_1 = 0;
    for(i=0;i<red.nnodos;i++){
        for(j=0;j<red.nodo[i].grado;j++){
            d_arista_2=0;
            for(k=0;k<red.nnodos;k++){
                for(l=0;l<red.nodo[k].grado;l++){
                        matriz[d_arista_1+d_arista_2] = \
                            kronecker(red.nodo[k].id,red.nodo[i].arista[j].entrada)* \
                            (1-kronecker(red.nodo[i].id,red.nodo[k].arista[l].entrada));
                    d_arista_2++;
                }
            }
            d_arista_1++;
        }
    }
}
    

int matriz(int argc, char *argv[])
{
    int u,i,j,k,l;

    fprintf(stderr,"Leyendo la red...\n");

    leer_red(&red,stdin);
    for (u=twom=0; u<h_red.nnodos; u++) twom += h_red.nodo[u].grado;
    fprintf(stderr,"Red con %i nodos y %i aristas\n",
        h_red.nnodos,twom/2);
    fprintf(stderr,"\n");

    FillStructs();

    int ** nbm;
    int renglon, columna;

    nbm = (int **)calloc(2*twom,sizeof(int));
    for (renglon=0; renglon<2*twom;renglon++){
        nbm[renglon] = (int *)calloc(2*twom,sizeof(int));
    }

    int (*d_nbm)[2*twom];

    hipMalloc((void**)&d_nbm, (4*twom*twom)*sizeof(int));

    hipMemcpy(d_nbm, nbm, (4*twom*twom)*sizeof(int), hipMemcpyHostToDevice);

    int *d_i,*d_j,*d_k,*d_l;

    hipMalloc((void**)&d_i,sizeof(int));
    hipMalloc((void**)&d_j,sizeof(int));
    hipMalloc((void**)&d_k,sizeof(int));
    hipMalloc((void**)&d_l,sizeof(int));

    hipMemcpy(d_i,&i,sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(d_j,&j,sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(d_k,&k,sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(d_l,&l,sizeof(int),hipMemcpyHostToDevice);

    generar<<<128,128>>>(*d_i,*d_j,*d_k,*d_l,*d_nbm);

    hipMemcpy (nbm, d_nbm, (4*twom*twom)*sizeof(int), hipMemcpyDeviceToHost);

    for(renglon=0;renglon<twom;renglon++){
        for(columna=0;columna<twom;columna++){
            printf("%i    ", nbm[renglon][columna]);
        }
    printf("\n");
    }
 
    hipFree(d_nbm);
    hipFree(d_i);
    hipFree(d_j);
    hipFree(d_k);
    hipFree(d_l);

    return 0;

}    
