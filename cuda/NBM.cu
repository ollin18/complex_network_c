#include "hip/hip_runtime.h"
#define VERBOSE

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>

#include "leergml.h"


RED red;
int twom;                // Va a ser el doble del número de aristas
                         // lo queremos para muchas cosas.
int **m;                 // El conteo de las aristas.

int arista_1, arista_2;

__host__ __device__ int kronecker(int x, int y){
    if(x==y) return 1;
    else return 0;
}



__global__ void generar(int i, int j, int k, int l, int * matriz){
    int d_arista_1 = threadIdx.x;
    int d_arista_2 = threadIdx.y;
    d_arista_1 = 0;
    for(i=0;i<red.nnodos;i++){
        for(j=0;j<red.nodo[i].grado;j++){
            d_arista_2=0;
            for(k=0;k<red.nnodos;k++){
                for(l=0;l<red.nodo[k].grado;l++){
                        matriz[d_arista_1+d_arista_2] = \
                            kronecker(red.nodo[k].id,red.nodo[i].arista[j].entrada)* \
                            (1-kronecker(red.nodo[i].id,red.nodo[k].arista[l].entrada));
                    d_arista_2++;
                }
            }
            d_arista_1++;
        }
    }
}
    

int main(int argc, char *argv[])
{
    int u,i,j,k,l;

    #ifdef VERBOSE
    fprintf(stderr,"Leyendo la red...\n");
    #endif
    leer_red(&red,stdin);
    for (u=twom=0; u<red.nnodos; u++) twom += red.nodo[u].grado;
    #ifdef VERBOSE
    fprintf(stderr,"Red con %i nodos y %i aristas\n",
        red.nnodos,twom/2);
    #endif
    #ifdef VERBOSE
    fprintf(stderr,"\n");
    #endif

    #ifdef VERBOSE
    int ** nbm;
    int renglon, columna;

    nbm = (int **)calloc(2*twom,sizeof(int));
    for (renglon=0; renglon<2*twom;renglon++){
        nbm[renglon] = (int *)calloc(2*twom,sizeof(int));
    }

    int (*d_nbm)[2*twom];

    hipMalloc((void**)&d_nbm, (4*twom*twom)*sizeof(int));

    hipMemcpy(d_nbm, nbm, (4*twom*twom)*sizeof(int), hipMemcpyHostToDevice);

    int *d_i,*d_j,*d_k,*d_l;

    hipMalloc((void**)&d_i,sizeof(int));
    hipMalloc((void**)&d_j,sizeof(int));
    hipMalloc((void**)&d_k,sizeof(int));
    hipMalloc((void**)&d_l,sizeof(int));

    hipMemcpy(d_i,&i,sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(d_j,&j,sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(d_k,&k,sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(d_l,&l,sizeof(int),hipMemcpyHostToDevice);

    generar<<<2,4>>>(*d_i,*d_j,*d_k,*d_l,*d_nbm);

    for(renglon=0;renglon<twom;renglon++){
        for(columna=0;columna<twom;columna++){
            printf("%i    ", nbm[renglon][columna]);
        }
    printf("\n");
    }
 
    #endif
    hipFree(d_nbm);
    hipFree(d_i);
    hipFree(d_j);
    hipFree(d_k);
    hipFree(d_l);

    return 0;

}    
